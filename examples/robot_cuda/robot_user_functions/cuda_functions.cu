#include "hip/hip_runtime.h"
/*!
********************************************************************************
********************************* CUDA SESSION *********************************
******************* Camilo A. Carvalho & Alberto F. De Souza *******************
********************** {camilo, alberto}@lcad.inf.ufes.br **********************
*********************************** AGO/2009 ***********************************
********************************************************************************
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "mae.h"
#include "filter.h"
#include "cuda_functions.h"


const int BLOCKSIZE = 64;


/*!
Copias de dados - de um NEURON_VECTOR para vetor float simples, e vice-versa
*/
extern "C" void
copyneuron2floatvec (float *float_vec, NEURON *neuron_vec, int size)
{
	int i;

	for (i = 0; i < size; i++)
		float_vec[i] = neuron_vec[i].output.fval;

	return;
}

extern "C" void
copyfloatvec2neuron (NEURON *neuron_vec, float *float_vec, int size)
{
	int i;

	for (i = 0; i < size; i++)
		neuron_vec[i].output.fval = float_vec[i];

	return;
}

/*!
Soma em arvore - retornar um valor simples de uma soma de elementos de um vetor
*/
__device__ void 
sum_tree_like_reduction (float *v, int size)
{
	int j, k;
	j=threadIdx.x;
	for(k = size / 2; k > 0; k >>= 1)
	{
		__syncthreads();

		v[j] += ( j < k )? v[k + j] : 0;
			
	}
}

/*!
Aloca as areas de memoria dos elementos de um PRIVATE_STATE na GPU
*/
extern "C" void 
cuda_alloc_device_data_structures (V1_MT_PRIVATE_STATE *v1_mt_private_state, int wi, int hi, int num_neurons)
{
	int dimension = wi*hi;

	hipMalloc ((void **) &(v1_mt_private_state->d_image_vector), dimension * sizeof (int));
	hipMalloc ((void **) &(v1_mt_private_state->d_xi), num_neurons * sizeof (int));
	hipMalloc ((void **) &(v1_mt_private_state->d_yi), num_neurons * sizeof (int));

	hipMalloc ((void **) &(v1_mt_private_state->d_s_r), num_neurons * sizeof (float));
	hipMalloc ((void **) &(v1_mt_private_state->d_s_r_q), num_neurons * sizeof (float));
	hipMalloc ((void **) &(v1_mt_private_state->d_s_l), num_neurons * sizeof (float));
	hipMalloc ((void **) &(v1_mt_private_state->d_s_l_q), num_neurons * sizeof (float));

	hipMalloc ((void **) &(v1_mt_private_state->d_mt), num_neurons * sizeof (float));

	hipMalloc ((void **) &(v1_mt_private_state->d_mt_gaussian), num_neurons * sizeof (float));
}



/*!
*********************************************************************************
* Function: map_v1
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel MAP_V1
__global__ void
map_v1 (int *xi, int *yi, int wi, int hi, int w, int h, int x_center, int y_center, float correction, float log_factor, int shift)
{
	float CUDA_LOG_POLAR_SCALE_FACTOR = 1.0;
	float CUDA_LOG_POLAR_THETA_CORRECTION = 0.0;
	float d, theta, exp_val, x;
	int i;

	int u = 0, v = 0;
	int adjust;

	for (v = blockIdx.x; v < h; v += gridDim.x)
	{
		for (u = threadIdx.x; u < w; u += blockDim.x)
		{
			i = v * w + u;

			if (u < w/2)
			{
				adjust = (w-1)/2 - u;
				x = ((float) adjust / (float) (w/2)) * log_factor;
				exp_val = (float) (wi/2) * (exp (log (log_factor) * (x - log_factor) / log_factor) - (1.0/log_factor)) * (log_factor / (log_factor - 1.0));
				d = CUDA_LOG_POLAR_SCALE_FACTOR * exp_val;
				theta = pi * ((h * (3.0 / 2.0) - (v * correction)) / h) + CUDA_LOG_POLAR_THETA_CORRECTION;
			}
			else
			{
				adjust = u - w/2;
				x = ((float) adjust / (float) (w/2)) * log_factor;
				exp_val = (float) (wi/2) * (exp (log (log_factor) * (x - log_factor) / log_factor) - (1.0/log_factor)) * (log_factor / (log_factor - 1.0));
				d = CUDA_LOG_POLAR_SCALE_FACTOR * exp_val;
				theta = pi * ((h * (3.0 / 2.0) + (v * correction)) / h) + CUDA_LOG_POLAR_THETA_CORRECTION;
			}

			xi[i] = (int) (d * cos(theta) + 0.5) + x_center + shift;
			yi[i] = (int) (d * sin(theta) + 0.5) + y_center;
		}
	}
}

// O filtro CUDA_MAP_V1_TO_IMAGE eh a chamada de kernel do filtro V1_MT
extern "C" void 
cuda_map_v1_to_image (int *xi, int *yi, int wi, int hi, int w, int h, int x_center, int y_center, float correction, float log_factor, int shift)
{
	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid (((w*h)/dimBlock.x) + (!((w*h)%dimBlock.x)?0:1));

	map_v1 <<< dimGrid , dimBlock >>> (xi, yi, wi, hi, w, h, x_center, y_center, correction, log_factor, shift);
}



/*!
*********************************************************************************
* Function: cuda_biological_gabor_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Alocacao de areas na memoria de constantes dos vetores dos campos receptivos
__constant__ int d_rf_points_x[256];
__constant__ int d_rf_points_y[256];
__constant__ float d_rf_points_w[256];
__constant__ float d_rf_points_g[256];

// Device CUDA_BIDIMENTIONAL_CONVOLUTION
__device__ float 
cuda_bidimentional_convolution (int rf_num_points, int *image_vector, int x_center, int y_center, int w, int h,
				float p_global_factor, float p_teste)
{
	int x_current, y_current, i, pixel;
	float intensity, red, green, blue;
	float accumulator=0.0f;

	// Initializes the accumulator variable
	//accumulator[threadIdx.x] = 0.0;

        for (i = 0; i < rf_num_points; i ++)
	{
		// Calculates the current point
		x_current = x_center + d_rf_points_x[i];
		y_current = y_center + d_rf_points_y[i];

		// Verifies if the point is inside of the neuron layer bounds 
		if ((x_current < 0) || (x_current >= w) || (y_current < 0) || (y_current >= h))
			continue;

		// Gets the output pixel value 
		pixel = image_vector[y_current * w + x_current];

		// Extracts the red, green and blue components of the pixel
		red   = (float) RED   (pixel);
		green = (float) GREEN (pixel);
		blue  = (float) BLUE  (pixel);

		// Calculates the intensity value
		intensity = p_teste + (p_global_factor * (red + green + blue)) / 3.0;

		// Accumulates the weighed intensity. The weight function depends of the position inside the kernel
		accumulator += d_rf_points_w[i] * d_rf_points_g[i] * intensity;
	}

	//sum_tree_like_reduction (accumulator, BLOCKSIZE);

	//__syncthreads();

	return (accumulator);
}


// Kernel CUDA_BIOLOGICAL_GABOR
__global__ void
cuda_biological_gabor (float *s_m, int *xi, int *yi, int wi, int hi, int size, int rf_num_points, int *image_vector,
			float global_factor, float c_delta_area, float teste)
{
	//int offset = gridDim.x*blockDim.x;
	int i = blockDim.x*blockIdx.x+threadIdx.x;
	
	//for (int i = tid; i < size; i += offset)
	if(i<size)
	{
		s_m[i] = c_delta_area * cuda_bidimentional_convolution (rf_num_points, image_vector, xi[i], yi[i],
									wi, hi, global_factor, teste) / 6.0;
	}
}



// O filtro CUDA_BIOLOGICAL_GABOR_NLS eh a chamada de kernel do filtro V1_MT
extern "C" void 
cuda_biological_gabor_nls (float *d_s_m, int *d_xi, int *d_yi, int *d_image_vector, int *xi, int *yi, int wi, int hi,
				int num_neurons, int rf_num_points, int *rf_points_x, int *rf_points_y, int *image_vector,
				float *rf_points_w, float *rf_points_g, float global_factor, float c_delta_area, float teste)
{
	int dimension = wi*hi;

	hipMemcpy (d_image_vector, image_vector, dimension * sizeof (int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w), rf_points_w, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, rf_num_points * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));

	cuda_biological_gabor <<< dimGrid , dimBlock >>> (d_s_m, d_xi, d_yi, wi, hi, num_neurons, rf_num_points, d_image_vector,
								global_factor, c_delta_area, teste);

	return;
}


// O filtro CUDA_BIOLOGICAL_GABOR_NLS_OLD so pode ser chamado individualmente, na estrutura original
extern "C" void 
cuda_biological_gabor_nls_old (float *s_m, int *xi, int *yi, int wi, int hi, int num_neurons, int rf_num_points,
				int *rf_points_x, int *rf_points_y, int *image_vector, float *rf_points_w, float *rf_points_g,
				float global_factor, float c_delta_area, float teste)
{
	int  *d_xi, *d_yi, *d_image_vector;
	float *d_s_m;
	int dimension = wi*hi;

	hipMalloc ((void **) &d_image_vector, dimension * sizeof (int));
	hipMalloc ((void **) &d_xi, num_neurons * sizeof (int));
	hipMalloc ((void **) &d_yi, num_neurons * sizeof (int));

	hipMemcpy (d_image_vector, image_vector, dimension * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy (d_xi, xi, num_neurons * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy (d_yi, yi, num_neurons * sizeof (int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w), rf_points_w, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, rf_num_points * sizeof (float));

	hipMalloc ((void **) &d_s_m, num_neurons * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid (num_neurons);

	cuda_biological_gabor <<< dimGrid , dimBlock >>> (d_s_m, d_xi, d_yi, wi, hi, num_neurons, rf_num_points, d_image_vector,
								global_factor, c_delta_area, teste);

	hipMemcpy (s_m, d_s_m, num_neurons * sizeof (float), hipMemcpyDeviceToHost);

	hipFree (d_s_m);
	hipFree (d_image_vector);
	hipFree (d_xi);
	hipFree (d_yi);

	return;
}



/*!
*********************************************************************************
* Function: cuda_add_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_ADD
__global__ void
cuda_add (float *d_C, float *d_A, float *d_B, int size)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < size )
		d_C[idx] = d_A[idx] + d_B[idx];

}

// O filtro CUDA_ADD_NLS so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_add_nls (NEURON *C, NEURON *A, NEURON *B, int size)
{

	float *h_A, *h_B, *h_C;
	float *d_A, *d_B, *d_C;

	h_A = (float *) malloc (size * sizeof (float));
	h_B = (float *) malloc (size * sizeof (float));
	h_C = (float *) malloc (size * sizeof (float));

	hipMalloc ((void **) &d_A, size * sizeof (float));
	hipMalloc ((void **) &d_B, size * sizeof (float));
	hipMalloc ((void **) &d_C, size * sizeof (float));

	copyneuron2floatvec (h_A, A, size);
	copyneuron2floatvec (h_B, B, size);

	hipMemcpy (d_A, h_A, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_B, h_B, size * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((size/dimBlock.x) + (!(size%dimBlock.x)?0:1));

	cuda_add <<< dimGrid , dimBlock >>> (d_C, d_A, d_B, size);

	hipMemcpy(h_C, d_C, size * sizeof (float), hipMemcpyDeviceToHost);

	copyfloatvec2neuron (C, h_C, size);

	hipFree (d_A);
	hipFree (d_B);
	hipFree (d_C);

	return;
}



/*!
*********************************************************************************
* Function: cuda_mult_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_MULT_ADD
__global__ void
cuda_mult_add (float *d_C, float *d_A, float *d_B, int size)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < size )
		d_C[idx] = (d_A[idx] * d_A[idx]) + (d_B[idx] * d_B[idx]);

}

// O filtro CUDA_MULT_NLS so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_mult_nls (NEURON *C, NEURON *A, NEURON *B, int size)
{
	float *h_A, *h_B, *h_C;
	float *d_A, *d_B, *d_C;

	h_A = (float *) malloc (size * sizeof (float));
	h_B = (float *) malloc (size * sizeof (float));
	h_C = (float *) malloc (size * sizeof (float));

	hipMalloc ((void **) &d_A, size * sizeof (float));
	hipMalloc ((void **) &d_B, size * sizeof (float));
	hipMalloc ((void **) &d_C, size * sizeof (float));

	copyneuron2floatvec (h_A, A, size);
	copyneuron2floatvec (h_B, B, size);

	hipMemcpy (d_A, h_A, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_B, h_B, size * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((size/dimBlock.x) + (!(size%dimBlock.x)?0:1));

	cuda_mult_add <<< dimGrid , dimBlock >>> (d_C, d_A, d_B, size);

	hipMemcpy(h_C, d_C, size * sizeof (float), hipMemcpyDeviceToHost);

	copyfloatvec2neuron (C, h_C, size);

	hipFree (d_A);
	hipFree (d_B);
	hipFree (d_C);

	return;
}



/*!
*********************************************************************************
* Function: cuda_div_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_DIV
__global__ void
cuda_div (float *d_D, float *d_A, float *d_B, float *d_C, int size, float k_param)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < size )
		d_D[idx] = d_C[idx] / (d_A[idx] + d_B[idx] + k_param);

}

// O filtro CUDA_DIV_NLS so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_div_nls (NEURON *D, NEURON *A, NEURON *B, NEURON *C, int size, float k_param)
{
	float *h_A, *h_B, *h_C, *h_D;
	float *d_A, *d_B, *d_C, *d_D;

	h_A = (float *) malloc (size * sizeof (float));
	h_B = (float *) malloc (size * sizeof (float));
	h_C = (float *) malloc (size * sizeof (float));
	h_D = (float *) malloc (size * sizeof (float));

	hipMalloc ((void **) &d_A, size * sizeof (float));
	hipMalloc ((void **) &d_B, size * sizeof (float));
	hipMalloc ((void **) &d_C, size * sizeof (float));
	hipMalloc ((void **) &d_D, size * sizeof (float));

	copyneuron2floatvec (h_A, A, size);
	copyneuron2floatvec (h_B, B, size);
	copyneuron2floatvec (h_C, C, size);

	hipMemcpy (d_A, h_A, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_B, h_B, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_C, h_C, size * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((size/dimBlock.x) + (!(size%dimBlock.x)?0:1));

	cuda_div <<< dimGrid , dimBlock >>> (d_D, d_A, d_B, d_C, size, k_param);

	hipMemcpy (h_D, d_D, size * sizeof (float), hipMemcpyDeviceToHost);

	copyfloatvec2neuron (D, h_D, size);

	hipFree (d_A);
	hipFree (d_B);
	hipFree (d_C);
	hipFree (d_D);

	return;
}



/*!
*********************************************************************************
* Function: cuda_bigfilter_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_BIGFILTER
__global__ void
cuda_bigfilter (float *mt, float *s_r, float *s_r_q, float *s_l, float *s_l_q, int num_neurons, float k)
{
	float s_lr, s_lr_q, c_l, c_r, c_lr;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < num_neurons )
	{

		s_lr = s_l[idx] + s_r[idx];
		s_lr_q = s_l_q[idx] + s_r_q[idx];

		c_l = (s_l[idx] * s_l[idx]) + (s_l_q[idx] * s_l_q[idx]);
		c_r = (s_r[idx] * s_r[idx]) + (s_r_q[idx] * s_r_q[idx]);
		c_lr = (s_lr * s_lr) + (s_lr_q * s_lr_q);

		mt[idx] = c_lr / (c_l + c_r + k);
	}
}

// O filtro CUDA_BIGFILTER_NLS eh a chamada de kernel do filtro V1_MT
extern "C" void
cuda_bigfilter_nls (float *d_mt, float *d_s_r, float *d_s_r_q, float *d_s_l, float *d_s_l_q, int num_neurons, float k)
{
	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));

	cuda_bigfilter <<< dimGrid , dimBlock >>> (d_mt, d_s_r, d_s_r_q, d_s_l, d_s_l_q, num_neurons, k);

	return;
}

// O filtro CUDA_BIGFILTER_NLS_OLD so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_bigfilter_nls_old (float *mt, float *s_r, float *s_r_q, float *s_l, float *s_l_q, int num_neurons, float k)
{
	float *d_mt, *d_s_r, *d_s_r_q, *d_s_l, *d_s_l_q;

	hipMalloc ((void **) &d_mt, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_r, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_r_q, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_l, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_l_q, num_neurons * sizeof (float));

	hipMemcpy (d_s_r, s_r, num_neurons * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_s_r_q, s_r_q, num_neurons * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_s_l, s_l, num_neurons * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_s_l_q, s_l_q, num_neurons * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));

	cuda_bigfilter <<< dimGrid , dimBlock >>> (d_mt, d_s_r, d_s_r_q, d_s_l, d_s_l_q, num_neurons, k);

	hipMemcpy (mt, d_mt, num_neurons * sizeof (float), hipMemcpyDeviceToHost);

	hipFree (d_mt);
	hipFree (d_s_r);
	hipFree (d_s_r_q);
	hipFree (d_s_l);
	hipFree (d_s_l_q);

	return;
}



/*!
*********************************************************************************
* Function: cuda_gaussian_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Device CUDA_GAUSSIAN
// __device__ float
// cuda_apply_gaussian_kernel (float *mt, int num_points, int x0, int y0, int wi, int hi)
// {
// 	int i, j, k;
// 	int xr, yr;
// 	float fltWeight;
// 
// 	__shared__ float fltWeightSum[BLOCKSIZE];
// 	__shared__ float fltResult[BLOCKSIZE];
// 
// 	fltWeightSum[threadIdx.x] = 0.0f;
// 	fltResult[threadIdx.x] = 0.0f;
// 
// 	for (i = threadIdx.x; i < num_points; i += blockDim.x)
// 	{
// 		xr = x0 + d_rf_points_x[i];
// 		yr = y0 + d_rf_points_y[i];
// 
// 		if ((xr < 0) || (xr >= wi) || (yr < 0) || (yr >= hi))
// 			continue;
// 
// 		fltWeightSum[threadIdx.x] += fltWeight = d_rf_points_g[i];
// 		fltResult[threadIdx.x] += fltWeight * mt[yr * wi + xr];
// 	}
// 
// 	for (k = BLOCKSIZE / 2; k > 0; k >>=1)
// 	{
// 		__syncthreads();
// 
// 		for (j = threadIdx.x; j < k; j += blockDim.x)
// 		{
// 			fltWeightSum[j] += fltWeightSum[k + j];
// 			fltResult[j] += fltResult[k + j];
// 		}
// 	}
// 
// 	__syncthreads();
// 
// 	return ((fltWeightSum[0] != .0f) ? fltResult[0] / fltWeightSum[0] : .0f);
// }

__device__ float
cuda_apply_gaussian_kernel (float *mt, int num_points, int x0, int y0, int wi, int hi)
{
	int i, k;
	int xr, yr;
	float fltWeight;

	__shared__ float fltWeightSum[BLOCKSIZE];
	__shared__ float fltResult[BLOCKSIZE];

	fltWeightSum[threadIdx.x] = 0.0f;
	fltResult[threadIdx.x] = 0.0f;

	for (i = threadIdx.x; i < num_points; i += blockDim.x)
	{
		xr = x0 + d_rf_points_x[i];
		yr = y0 + d_rf_points_y[i];

		if ((xr < 0) || (xr >= wi) || (yr < 0) || (yr >= hi))
			continue;

		fltWeightSum[threadIdx.x] += fltWeight = d_rf_points_g[i];
		fltResult[threadIdx.x] += fltWeight * mt[yr * wi + xr];
	}
	i = threadIdx.x;
	for (k = BLOCKSIZE / 2; k > 0; k >>=1)
	{
		__syncthreads();

		if( i < k )
		{
			fltWeightSum[i] += fltWeightSum[k + i];
			fltResult[i] += fltResult[k + i];
		}
	}

	__syncthreads();

	return ((fltWeightSum[0] != .0f) ? fltResult[0] / fltWeightSum[0] : .0f);
}

// Kernel CUDA_GAUSSIAN
__global__ void
cuda_gaussian (float *mt_gaussian, float *mt, int num_points, int wo, int ho, int wi, int hi)
{
	for (int i = blockIdx.x; i < wo; i += gridDim.x)
	{
		for (int j = blockIdx.y; j < ho; j += gridDim.y)
		{
			mt_gaussian[j * wo + i] = cuda_apply_gaussian_kernel (mt, num_points, i, j, wo, ho);
		}
	}
}

// O filtro CUDA_GAUSSIAN_NLS eh a chamada de kernel do filtro V1_MT
extern "C" void
cuda_gaussian_nls (float *mt_gaussian, float *d_mt_gaussian, float *d_mt, int *rf_points_x, int *rf_points_y, 
		  float *rf_points_g, int num_points, int wo, int ho, int wi, int hi)
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, num_points * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid (wo,ho);

	cuda_gaussian <<< dimGrid , dimBlock >>> (d_mt_gaussian, d_mt, num_points, wo, ho, wi, hi);

	hipMemcpy (mt_gaussian, d_mt_gaussian, wo*ho * sizeof (float), hipMemcpyDeviceToHost);
}

// O filtro CUDA_GAUSSIAN_NLS_OLD so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_gaussian_nls_old (float *mt_gaussian, float *mt, int *rf_points_x, int *rf_points_y, 
		  float *rf_points_g, int num_points, int wo, int ho, int wi, int hi)
{
	float *d_mt, *d_mt_gaussian;

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, num_points * sizeof (float));

	hipMalloc ((void **) &d_mt, wi*hi * sizeof (float));
	hipMemcpy (d_mt, mt, wi*hi * sizeof (float), hipMemcpyHostToDevice);

	hipMalloc ((void **) &d_mt_gaussian, wo*ho * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid (wo,ho);

	cuda_gaussian <<< dimGrid , dimBlock >>> (d_mt_gaussian, d_mt, num_points, wo, ho, wi, hi);

	hipMemcpy (mt_gaussian, d_mt_gaussian, wo*ho * sizeof (float), hipMemcpyDeviceToHost);

	hipFree (d_mt_gaussian);
	hipFree (d_mt);
}

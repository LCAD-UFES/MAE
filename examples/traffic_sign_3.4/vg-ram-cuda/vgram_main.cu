#include "hip/hip_runtime.h"
/**
 * Controle de versão: 3.0
 *
 * Esta versão foi modificada a partir da versão 1.1.
 * I - PATTERN_UNIT_SIZE estava especificado em bytes e nao em bits, agora este
 * valor está especificado em bits... mas eh dividido por 8 nos mallocs, pois estes ainda trabalham em bytes.
 * II - A quase totalidade dos int's foi alterada para unsigned int, uma vez
 * que inteiros com sinal sofrem o Shift aritimetico (http://www.cs.uaf.edu/~cs301/notes/Chapter5/node3.html)
 * quando deslocados para a direita em vez do Shift logico, que era o esperado.
 * III - A taxa de acerto da RN para o Linux permanece a mesma, porem eh alterada para o Windows.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <cutil.h>

#include "vgram.h"
#include "vgram_files.h"

// Variaveis globais //
VG_RAM_WNN cuda_vg_ram_wnn;
VG_RAM_WNN host_vg_ram_wnn;

DATA_SET cuda_training_set;
DATA_SET host_training_set;

DATA_SET cuda_testing_set;
DATA_SET host_testing_set;

int *Rand_h, *Rand_d;
int *candidates_d;
int *bit_pattern_d;

// Código //

__device__ inline void
cudaSetNeuronMemory(int *mem, int training_set_num_samples, int memory_bit_group_size, int neuron, int sample, int value)
{
	mem[(neuron * training_set_num_samples + sample) * (memory_bit_group_size+1) + memory_bit_group_size] = value;
}

__device__ inline int
cudaGetNeuronMemory(int *mem, int training_set_num_samples, int memory_bit_group_size, int neuron, int sample)
{
	return mem[(neuron * training_set_num_samples + sample) * (memory_bit_group_size+1) + memory_bit_group_size];
}

__device__ inline int *
cudaGetNeuronMemoryByNeuronAndSample(int *mem, int training_set_num_samples, int memory_bit_group_size, int neuron, int sample)
{
	return &mem[(neuron * training_set_num_samples + sample) * (memory_bit_group_size+1)];
}

__device__ inline int *
cudaGetNeuronMemoryByNeuron(int *mem, int training_set_num_samples, int memory_bit_group_size, int neuron)
{
	return &mem[neuron * training_set_num_samples * (memory_bit_group_size+1)];
}

__device__ inline int *
cudaGetNeuronMemoryBySample(int *mem, int memory_bit_group_size, int sample)
{
	return &mem[sample * (memory_bit_group_size+1)];
}

void
cudaAllocDataSet(DATA_SET *data_set, int num_samples, int num_inputs)
{
	CUDA_SAFE_CALL(hipMalloc((void **) &(data_set->sample_class), num_samples * sizeof(int)));
#ifdef GREYSCALE
	CUDA_SAFE_CALL(hipMalloc((void **) &(data_set->sample), num_inputs * num_samples * sizeof(float)));
#else
	CUDA_SAFE_CALL(hipMalloc((void **) &(data_set->sample), num_inputs * num_samples * sizeof(int)));
#endif
}


void
cudaDeallocateDataSet(DATA_SET *data_set)
{
	hipFree(data_set->sample);

	hipFree(data_set->sample_class);
}

int
cudaLoadDataSet(DATA_SET *data_set_to, const DATA_SET *data_set_from, int num_inputs)
{
	int num_samples = data_set_from->num_samples;

	cudaAllocDataSet(data_set_to, num_samples, num_inputs);

	CUDA_SAFE_CALL(hipMemcpy(data_set_to->sample_class, data_set_from->sample_class, num_samples * sizeof(int), hipMemcpyHostToDevice));
#ifdef GREYSCALE
	CUDA_SAFE_CALL(hipMemcpy(data_set_to->sample, data_set_from->sample, num_samples * num_inputs * sizeof(float), hipMemcpyHostToDevice));
#else
	CUDA_SAFE_CALL(hipMemcpy(data_set_to->sample, data_set_from->sample, num_samples * num_inputs * sizeof(int), hipMemcpyHostToDevice));
#endif

	return num_samples;
}


void
LoadTrainingSet(char *file_name)
{
	host_training_set.num_samples = LoadDataSet(file_name, &host_training_set, host_vg_ram_wnn.network_input_size);
	cuda_training_set.num_samples = cudaLoadDataSet(&cuda_training_set, &host_training_set, host_vg_ram_wnn.network_input_size);

}


void
LoadTestingSet(char *file_name)
{
	host_testing_set.num_samples = LoadDataSet(file_name, &host_testing_set, host_vg_ram_wnn.network_input_size);
	cuda_testing_set.num_samples = cudaLoadDataSet(&cuda_testing_set, &host_testing_set, host_vg_ram_wnn.network_input_size);

}


void
AllocateNetworkSynapses()
{
	if ((host_vg_ram_wnn.synapses = (int *) malloc(host_vg_ram_wnn.number_of_neurons * host_vg_ram_wnn.number_of_synapses_per_neuron * sizeof(int))) == NULL)
		Error("Could not allocate memory in AllocateNetworkSynapses() for host_vg_ram_wnn.synapses.", "", "");
}

void
cudaAllocateNetworkSynapses()
{
	if (hipMalloc((void **) &(cuda_vg_ram_wnn.synapses), cuda_vg_ram_wnn.number_of_synapses_per_neuron * cuda_vg_ram_wnn.number_of_neurons * sizeof(int)) != hipSuccess)
		Error("Could not allocate memory in cudaAllocateNetworkSynapses() for vg_ram_wnn.synapses.", "", "");
}

void
AllocateNetworkMemories()
{
	if ((host_vg_ram_wnn.number_of_synapses_per_neuron % PATTERN_UNIT_SIZE) == 0)
		host_vg_ram_wnn.memory_bit_group_size = host_vg_ram_wnn.number_of_synapses_per_neuron / PATTERN_UNIT_SIZE;
	else
		host_vg_ram_wnn.memory_bit_group_size = 1 + host_vg_ram_wnn.number_of_synapses_per_neuron / PATTERN_UNIT_SIZE;

	if ((host_vg_ram_wnn.memories = (int *) malloc(host_vg_ram_wnn.number_of_neurons * host_training_set.num_samples * (host_vg_ram_wnn.memory_bit_group_size + 1) * sizeof(int))) == NULL)
		Error("Could not allocate memory for memories in AllocateNetworkMemories()", "", "");
}

void
cudaAllocateNetworkMemories()
{
	if ((cuda_vg_ram_wnn.number_of_synapses_per_neuron % PATTERN_UNIT_SIZE) == 0)
		cuda_vg_ram_wnn.memory_bit_group_size = cuda_vg_ram_wnn.number_of_synapses_per_neuron / PATTERN_UNIT_SIZE;
	else
		cuda_vg_ram_wnn.memory_bit_group_size = 1 + cuda_vg_ram_wnn.number_of_synapses_per_neuron / PATTERN_UNIT_SIZE;

	if (hipMalloc((void**) &(cuda_vg_ram_wnn.memories), cuda_vg_ram_wnn.number_of_neurons * cuda_training_set.num_samples * (cuda_vg_ram_wnn.memory_bit_group_size + 1) * sizeof(int)) != hipSuccess)
		Error("Could not allocate memory for memories in cudaAllocateNetworkMemories()", "", "");
}

void
AllocateNetworkOutput()
{
	if ((host_vg_ram_wnn.neuron_output = (int *) malloc(host_testing_set.num_samples * host_vg_ram_wnn.number_of_neurons * sizeof(int))) == NULL)
		Error("Could not allocate memory for the neurons output in AllocateNetworkOutput().", "", "");
}

void
cudaAllocateNetworkOutput()
{
	if (hipMalloc((void**) &(cuda_vg_ram_wnn.neuron_output), cuda_testing_set.num_samples * cuda_vg_ram_wnn.number_of_neurons * sizeof(int)) != hipSuccess)
		Error("Could not allocate memory for the neurons output in cudaAllocateNetworkOutput().", "", "");
}

void
cudaLoadSynapticInterconnectionPattern()
{
	CUDA_SAFE_CALL(hipMemcpy(cuda_vg_ram_wnn.synapses, host_vg_ram_wnn.synapses, host_vg_ram_wnn.number_of_neurons * host_vg_ram_wnn.number_of_synapses_per_neuron * sizeof(int), hipMemcpyHostToDevice));
}

void
cudaLoadNetworkConfiguration(const VG_RAM_WNN *host_vg_ram_wnn, VG_RAM_WNN *cuda_vg_ram_wnn)
{
	cuda_vg_ram_wnn->number_of_neurons = host_vg_ram_wnn->number_of_neurons;
	cuda_vg_ram_wnn->number_of_synapses_per_neuron = host_vg_ram_wnn->number_of_synapses_per_neuron;
	cuda_vg_ram_wnn->network_input_size = host_vg_ram_wnn->network_input_size;

}

void
BuildNetwork()
{
	AllocateNetworkSynapses();
	cudaAllocateNetworkSynapses();

	AllocateNetworkMemories();
	cudaAllocateNetworkMemories();
}

void
ClearNetwork()
{
//	TODO: liberar memória
//	DeallocateNetworkOutput();
//	cudaDeallocateNetworkOutput();
//
//	DeallocateNetworkSynapses();
//	cudaDeallocateNetworkSynapses();
//
//	DeallocateNetworkMemories();
//	cudaDeallocateNetworkMemories();
}

__device__ void
cudaBuildBitPattern(unsigned int *bit_pattern, int *synapses,
#ifdef GREYSCALE
		float *network_input,
#else
		int *network_input,
#endif
		int number_of_synapses_per_neuron)
{
	int synapse;
	int current_bit_pattern_group;
	int tid = threadIdx.x%PATTERN_UNIT_SIZE;
	__shared__ unsigned int bp[64];
	unsigned int aux;

	if(threadIdx.x<64) bp[threadIdx.x]=0;

	__syncthreads();

	for (synapse = threadIdx.x; synapse < number_of_synapses_per_neuron -1; synapse += blockDim.x)
	{
		// Cada grupo de padrão de bits tem tamanho igual a PATTERN_UNIT_SIZE.
		current_bit_pattern_group = synapse / PATTERN_UNIT_SIZE;

		// Minchington: Uma sinapse compara com a próxima
		aux = (network_input[synapses[synapse]] > network_input[synapses[synapse+1]]) ? 1 : 0;

		aux  = aux << tid;

		atomicOr(&bp[current_bit_pattern_group],aux);

	}
	__syncthreads();

	if(threadIdx.x == blockDim.x)
	{
		current_bit_pattern_group = synapse / PATTERN_UNIT_SIZE;
		aux  = aux << tid;
		aux = (network_input[synapses[synapse]] > network_input[synapses[0]]) ? 1 : 0;
		atomicOr(&bp[current_bit_pattern_group],aux);
		//atomicExch(&bit_pattern[current_bit_pattern_group], bp[current_bit_pattern_group]);

	}

	__syncthreads();

	if(threadIdx.x < number_of_synapses_per_neuron / PATTERN_UNIT_SIZE)
	{
		bit_pattern[threadIdx.x] =  bp[threadIdx.x];
	}


}

__global__ void
cudaNeuronTrain(VG_RAM_WNN cuda_vg_ram_wnn, DATA_SET cuda_training_set, int sample)
{
	int sample_class;
	cuda_vg_ram_wnn.network_input = &(cuda_training_set.sample[sample * cuda_vg_ram_wnn.network_input_size]);
	sample_class = cuda_training_set.sample_class[sample];
	for (int neuron = blockIdx.x; neuron < cuda_vg_ram_wnn.number_of_neurons; neuron += gridDim.x)
	{
		// Escreve diretamente na memória do neurônio o padrão de bits gerado a partir da sample
		cudaBuildBitPattern((unsigned int *)
				cudaGetNeuronMemoryByNeuronAndSample(
						cuda_vg_ram_wnn.memories,
						cuda_training_set.num_samples,
						cuda_vg_ram_wnn.memory_bit_group_size,
						neuron, sample),
							&(cuda_vg_ram_wnn.synapses[neuron * cuda_vg_ram_wnn.number_of_synapses_per_neuron]),
							cuda_vg_ram_wnn.network_input,
							cuda_vg_ram_wnn.number_of_synapses_per_neuron);

		// Escreve diretamente na memória do neurônio a classe associada ao padrão de bits
		cudaSetNeuronMemory(cuda_vg_ram_wnn.memories,
				cuda_training_set.num_samples,
				cuda_vg_ram_wnn.memory_bit_group_size,
				neuron, sample, sample_class);
	}

}

void
cudaTrain()
{
	for (int sample = 0; sample < cuda_training_set.num_samples; sample ++)
	{
		cudaNeuronTrain<<<1024,BLOCK_DIM>>>(cuda_vg_ram_wnn,cuda_training_set,sample);
		printf("Training sample %d de %d.\n", sample+1, cuda_training_set.num_samples);
	}
}


__device__ int
cudaBitcount(unsigned int n)
{
   unsigned int tmp;

   tmp = n - ((n >> 1) & 033333333333)
           - ((n >> 2) & 011111111111);
   return ((tmp + (tmp >> 3)) & 030707070707) % 63;
}

__device__ int
cudaHammingDistance(unsigned int *bit_pattern1, unsigned int *bit_pattern2,  int memory_bit_group_size)
{
	int i;
	unsigned int bit_difference;
	int hamming_distance;

	hamming_distance = 0;
	for (i = 0; i < memory_bit_group_size; i++)
	{
		bit_difference = bit_pattern1[i] ^ bit_pattern2[i];
		hamming_distance += cudaBitcount(bit_difference);
	}
	return hamming_distance;
}

__device__ void
cudaFindNearestPattern(unsigned int *bit_pattern,
					   int *neuron_memory,
					   int *neuron_output,
					   int training_set_num_samples,
					   int memory_bit_group_size)
{
	int hd_register;
	__shared__ int short_hd;

	int best_hd = 1<<30;
	int best_sample;
	short_hd = 1<<30;

	__syncthreads();

	for (int learned_pattern = threadIdx.x; learned_pattern < training_set_num_samples; learned_pattern += blockDim.x)
	{
		hd_register = cudaHammingDistance((unsigned int *)
				cudaGetNeuronMemoryBySample(neuron_memory, memory_bit_group_size, learned_pattern), bit_pattern, memory_bit_group_size);
		if(hd_register < best_hd)
		{
			best_hd = hd_register;
			best_sample = learned_pattern;
		}
	}

	atomicMin(&short_hd, best_hd);

	__syncthreads();

	if(short_hd == best_hd)
	{
		*neuron_output = cudaGetNeuronMemoryBySample(neuron_memory, memory_bit_group_size, best_sample)[memory_bit_group_size];
	}

}

__global__ void
cudaNeuronTest(VG_RAM_WNN cuda_vg_ram_wnn, DATA_SET cuda_training_set, DATA_SET cuda_testing_set, int sample)
{
	__shared__
	unsigned int bit_pattern[64];

	cuda_vg_ram_wnn.network_input = &(cuda_testing_set.sample[sample * cuda_vg_ram_wnn.network_input_size]);

	for (int neuron = blockIdx.x; neuron < cuda_vg_ram_wnn.number_of_neurons; neuron += gridDim.x)
	{
		cudaBuildBitPattern(bit_pattern,
					&(cuda_vg_ram_wnn.synapses[neuron * cuda_vg_ram_wnn.number_of_synapses_per_neuron]),
					cuda_vg_ram_wnn.network_input, cuda_vg_ram_wnn.number_of_synapses_per_neuron);

		cudaFindNearestPattern(bit_pattern,
					cudaGetNeuronMemoryByNeuron(cuda_vg_ram_wnn.memories,
					cuda_training_set.num_samples,
					cuda_vg_ram_wnn.memory_bit_group_size,
					neuron),
				    &(cuda_vg_ram_wnn.neuron_output[sample * cuda_vg_ram_wnn.number_of_neurons + neuron]),
				    cuda_training_set.num_samples,
				    cuda_vg_ram_wnn.memory_bit_group_size);
	}
}


void
cudaTest()
{
	for (int sample = 0; sample < cuda_testing_set.num_samples; sample++)
	{
		cudaNeuronTest<<<1024,BLOCK_DIM>>>(cuda_vg_ram_wnn,cuda_training_set,cuda_testing_set,sample);
		printf("Testing sample %d de %d.\n", sample+1, cuda_testing_set.num_samples);
	}
}

int
main (int argc, char *argv[])
{
	if (argc != 5)
		Error("Wrong number of arguments.\n", "Usage: vg-ram-wnn.exe network-configuration.csv synaptic-interconnection-pattern.csv training-set.csv testing-set.csv\n", "");

	LoadNetworkConfiguration(argv[1], &host_vg_ram_wnn);
	cudaLoadNetworkConfiguration(&host_vg_ram_wnn, &cuda_vg_ram_wnn);

	LoadTrainingSet(argv[3]);

	BuildNetwork();

	LoadSynapticInterconnectionPattern(argv[2], &host_vg_ram_wnn);
	cudaLoadSynapticInterconnectionPattern();

	cudaTrain();

	DeallocateDataSet(&host_training_set);
	cudaDeallocateDataSet(&cuda_training_set);

//	CUDA_SAFE_CALL(hipMemcpy(host_vg_ram_wnn.memories, cuda_vg_ram_wnn.memories,
//			cuda_vg_ram_wnn.number_of_neurons * cuda_training_set.num_samples * (cuda_vg_ram_wnn.memory_bit_group_size + 1) * sizeof(int),
//			hipMemcpyDeviceToHost));
//
//	Save(&host_vg_ram_wnn, &host_training_set, "memory.log");

	printf("Fim do treinamento.\n");

	LoadTestingSet(argv[4]);

	AllocateNetworkOutput();
	cudaAllocateNetworkOutput();

	cudaTest();

	CUDA_SAFE_CALL(hipMemcpy(host_vg_ram_wnn.neuron_output, cuda_vg_ram_wnn.neuron_output,
			cuda_testing_set.num_samples * cuda_vg_ram_wnn.number_of_neurons * sizeof(int),
			hipMemcpyDeviceToHost));

	EvaluateNetworkOutputs(&host_vg_ram_wnn, &host_testing_set);

	DeallocateDataSet(&host_testing_set);
	cudaDeallocateDataSet(&cuda_testing_set);

	ClearNetwork();

	printf("Program finished OK!");

	return 0;
}

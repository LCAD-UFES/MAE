#include "hip/hip_runtime.h"
/*!
********************************************************************************
********************************* CUDA SESSION *********************************
******************* Camilo A. Carvalho & Alberto F. De Souza *******************
********************** {camilo, alberto}@lcad.inf.ufes.br **********************
*********************************** AGO/2009 ***********************************
********************************************************************************
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "mae.h"
#include "filter.h"
#include "cuda_functions.h"


const int BLOCKSIZE = 64;


/*!
Copias de dados - de um NEURON_VECTOR para vetor float simples, e vice-versa
*/
extern "C" void
copyneuron2floatvec (float *float_vec, NEURON *neuron_vec, int size)
{
	int i;

	for (i = 0; i < size; i++)
		float_vec[i] = neuron_vec[i].output.fval;

	return;
}

extern "C" void
copyfloatvec2neuron (NEURON *neuron_vec, float *float_vec, int size)
{
	int i;

	for (i = 0; i < size; i++)
		neuron_vec[i].output.fval = float_vec[i];

	return;
}

/*!
Soma em arvore - retornar um valor simples de uma soma de elementos de um vetor
*/
__device__ void 
sum_tree_like_reduction (float *v, int size)
{
	int j, k;
	j=threadIdx.x;
	for(k = size / 2; k > 0; k >>= 1)
	{
		__syncthreads();
		if(j < k)
			v[j] += v[k + j];
	}
}

/*!
Aloca as areas de memoria dos elementos de um PRIVATE_STATE na GPU
*/
extern "C" void 
cuda_alloc_device_data_structures (V1_MT_PRIVATE_STATE *v1_mt_private_state, int wi, int hi, int num_neurons)
{
	int dimension = wi*hi;

	hipMalloc ((void **) &(v1_mt_private_state->d_image_vector), dimension * sizeof (int));
	hipMalloc ((void **) &(v1_mt_private_state->d_xi), num_neurons * sizeof (int));
	hipMalloc ((void **) &(v1_mt_private_state->d_yi), num_neurons * sizeof (int));

	hipMalloc ((void **) &(v1_mt_private_state->d_s_r), num_neurons * sizeof (float));
	hipMalloc ((void **) &(v1_mt_private_state->d_s_r_q), num_neurons * sizeof (float));
	hipMalloc ((void **) &(v1_mt_private_state->d_s_l), num_neurons * sizeof (float));
	hipMalloc ((void **) &(v1_mt_private_state->d_s_l_q), num_neurons * sizeof (float));

	hipMalloc ((void **) &(v1_mt_private_state->d_mt), num_neurons * sizeof (float));

	hipMalloc ((void **) &(v1_mt_private_state->d_mt_gaussian), num_neurons * sizeof (float));

    hipMalloc ((void **) &(v1_mt_private_state->I_translated_image_left), num_neurons * sizeof (int));
}



/*!
*********************************************************************************
* Function: map_v1
* Description:
* Inputs:
* Output:
*********************************************************************************
*/




/*!
*********************************************************************************
* Function: cuda_biological_gabor_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Alocacao de areas na memoria de constantes dos vetores dos campos receptivos
__constant__ int d_rf_points_x[256];
__constant__ int d_rf_points_y[256];
__constant__ float d_rf_points_w[256];
__constant__ float d_rf_points_g[256];

__constant__ int d_rf_points_x_q[256];
__constant__ int d_rf_points_y_q[256];
__constant__ float d_rf_points_w_q[256];
__constant__ float d_rf_points_g_q[256];

// Device CUDA_BIDIMENTIONAL_CONVOLUTION

__device__ void
cuda_bidimentional_convolution_msq (int rf_num_points, int *image_vector, int x_center, int y_center, int w, int h,
				float p_global_factor, float p_teste,float *output,float *output_q)
{
	int x_current, y_current, i, pixel;
	float intensity, red, green, blue;
	float accumulator=0.0f;
	float accumulator_q=0.0f;

	// Initializes the accumulator variable
	//accumulator[threadIdx.x] = 0.0;

        for (i = 0; i < rf_num_points; i ++)
	{
		// Calculates the current point
		x_current = x_center + d_rf_points_x[i];
		y_current = y_center + d_rf_points_y[i];
		
		// Verifies if the point is inside of the neuron layer bounds 
		if (!((x_current < 0) || (x_current >= w) || (y_current < 0) || (y_current >= h))){
			

			// Gets the output pixel value 
			pixel = image_vector[y_current * w + x_current];

			// Extracts the red, green and blue components of the pixel
			red   = (float) RED   (pixel);
			green = (float) GREEN (pixel);
			blue  = (float) BLUE  (pixel);

			// Calculates the intensity value
			intensity = p_teste + (p_global_factor * (red + green + blue)) / 3.0f;

			// Accumulates the weighed intensity. The weight function depends of the position inside the kernel
			accumulator += d_rf_points_w[i] * d_rf_points_g[i] * intensity;
		}
		
		x_current = x_center + d_rf_points_x_q[i];
		y_current = y_center + d_rf_points_y_q[i];
		
		// Verifies if the point is inside of the neuron layer bounds 
		if ((x_current < 0) || (x_current >= w) || (y_current < 0) || (y_current >= h))
			continue;

		// Gets the output pixel value 
		pixel = image_vector[y_current * w + x_current];

		// Extracts the red, green and blue components of the pixel
		red   = (float) RED   (pixel);
		green = (float) GREEN (pixel);
		blue  = (float) BLUE  (pixel);

		// Calculates the intensity value
		intensity = p_teste + (p_global_factor * (red + green + blue)) / 3.0f;

		// Accumulates the weighed intensity. The weight function depends of the position inside the kernel
		accumulator_q += d_rf_points_w_q[i] * d_rf_points_g_q[i] * intensity;
	}
	
	*output = accumulator;
	*output_q = accumulator_q;

	//sum_tree_like_reduction (accumulator, BLOCKSIZE);

	//__syncthreads();

	//return (accumulator);
}

/*
__device__ void
cuda_bidimentional_convolution_msq (int rf_num_points, int *image_vector, int x_center, int y_center, int w, int h,
				float p_global_factor, float p_teste,float *output,float *output_q)
{
	int x_current, y_current, i, pixel;
	float intensity, red, green, blue;
	__shared__ float accumulator[BLOCKSIZE];
	__shared__ float accumulator_q[BLOCKSIZE];

	// Initializes the accumulator variable
	accumulator[threadIdx.x] = 0.0f;
	accumulator_q[threadIdx.x] = 0.0f;

    for (i = threadIdx.x; i < rf_num_points; i +=blockDim.x)
	{
		// Calculates the current point
		x_current = x_center + d_rf_points_x[i];
		y_current = y_center + d_rf_points_y[i];
		
		// Verifies if the point is inside of the neuron layer bounds 
		if (!((x_current < 0) || (x_current >= w) || (y_current < 0) || (y_current >= h))){
			

			// Gets the output pixel value 
			pixel = image_vector[y_current * w + x_current];

			// Extracts the red, green and blue components of the pixel
			red   = (float) RED   (pixel);
			green = (float) GREEN (pixel);
			blue  = (float) BLUE  (pixel);

			// Calculates the intensity value
			intensity = p_teste + (p_global_factor * (red + green + blue)) / 3.0f;

			// Accumulates the weighed intensity. The weight function depends of the position inside the kernel
			accumulator[threadIdx.x]  += d_rf_points_w[i] * d_rf_points_g[i] * intensity;
		}
		
		x_current = x_center + d_rf_points_x_q[i];
		y_current = y_center + d_rf_points_y_q[i];
		
		// Verifies if the point is inside of the neuron layer bounds 
		if ((x_current < 0) || (x_current >= w) || (y_current < 0) || (y_current >= h))
			continue;

		// Gets the output pixel value 
		pixel = image_vector[y_current * w + x_current];

		// Extracts the red, green and blue components of the pixel
		red   = (float) RED   (pixel);
		green = (float) GREEN (pixel);
		blue  = (float) BLUE  (pixel);

		// Calculates the intensity value
		intensity = p_teste + (p_global_factor * (red + green + blue)) / 3.0f;

		// Accumulates the weighed intensity. The weight function depends of the position inside the kernel
		accumulator_q[threadIdx.x]  += d_rf_points_w_q[i] * d_rf_points_g_q[i] * intensity;
	}
	
	
	i=threadIdx.x;
	for(int k = BLOCKSIZE >> 2; k > 0; k >>= 1)
	{
		__syncthreads();
		if(i < k){
			accumulator[i] += accumulator[k + i];
			accumulator_q[i] += accumulator_q[k + i];
		}
	}
	
	//if(threadIdx.x==0){
		*output = accumulator[0];
		*output_q = accumulator_q[0];
	//}

	//sum_tree_like_reduction (accumulator, BLOCKSIZE);

	__syncthreads();

	//return (accumulator);
}*/

__device__ float 
cuda_bidimentional_convolution (int rf_num_points, int *image_vector, int x_center, int y_center, int w, int h,
				float p_global_factor, float p_teste)
{
	int x_current, y_current, i, pixel;
	float intensity, red, green, blue;
	float accumulator=0.0f;

	// Initializes the accumulator variable
	//accumulator[threadIdx.x] = 0.0;

        for (i = 0; i < rf_num_points; i ++)
	{
		// Calculates the current point
		x_current = x_center + d_rf_points_x[i];
		y_current = y_center + d_rf_points_y[i];

		// Verifies if the point is inside of the neuron layer bounds 
		if ((x_current < 0) || (x_current >= w) || (y_current < 0) || (y_current >= h))
			continue;

		// Gets the output pixel value 
		pixel = image_vector[y_current * w + x_current];

		// Extracts the red, green and blue components of the pixel
		red   = (float) RED   (pixel);
		green = (float) GREEN (pixel);
		blue  = (float) BLUE  (pixel);

		// Calculates the intensity value
		intensity = p_teste + (p_global_factor * (red + green + blue)) / 3.0f;

		// Accumulates the weighed intensity. The weight function depends of the position inside the kernel
		accumulator += d_rf_points_w[i] * d_rf_points_g[i] * intensity;
	}
	
	

	//sum_tree_like_reduction (accumulator, BLOCKSIZE);

	//__syncthreads();

	return (accumulator);
}

// cuda_biological_gabor (float *s_m,  
// 		       int wi, 
// 		       int hi, 
// 		       int size, 
// 		       int rf_num_points, int *image_vector,
// 			float global_factor, float c_delta_area, float teste, int wo, int ho,
// 			float min_dog)


__global__ void
cuda_biological_gabor_right (float *d_s_r, float *d_s_r_q,
				 int *image_vector, 
				 int wi, int hi, int size, 
				 int rf_num_points, 
				 float global_factor, 
				 float c_delta_area, float teste,int wo, int ho,
				 float min_dog,float min_dog_q)
{
	//int offset = gridDim.x*blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
		
	
	//for (int i = tid; i < size; i += offset) i = 
	if( i<size )
	{	
		int xi=i%wo;
		int yi=i/wo;
		float result,result_q;
		cuda_bidimentional_convolution_msq (rf_num_points, image_vector, xi, yi,wi, hi, global_factor, teste,&result,&result_q);
		//if(threadIdx.x==0){
			result = c_delta_area * result  / 6.0f;
			result_q = c_delta_area * result_q  / 6.0f;
			
			d_s_r[i] = (((result > 0.0f) && (result <= min_dog)) || ((result < 0.0f) && (result >= min_dog)))? 0.0f: result;
			d_s_r_q[i] = (((result_q > 0.0f) && (result_q <= min_dog_q)) || ((result_q < 0.0f) && (result_q >= min_dog_q)))? 0.0f: result_q;
		//}
		
		
	}
}

extern "C" void 
cuda_biological_gabor_nls_right (float *d_s_r, float *d_s_r_q,
				 int *d_image_vector, 
				 int wi, int hi, int num_neurons, 
				 int rf_num_points, 
				 int *rf_points_x, 
				 int *rf_points_y, 
				 int *rf_points_x_q, 
				 int *rf_points_y_q,
				 int *image_vector,
				 float *rf_points_w, 
				 float *rf_points_g, 
				 float *rf_points_w_q, 
				 float *rf_points_g_q, 
				 float global_factor, 
				 float c_delta_area, float teste,int wo, int ho,
				 float min_dog,float min_dog_q)
{
	int dimension = wi*hi;

	hipMemcpy (d_image_vector, image_vector, dimension * sizeof (int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w), rf_points_w, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, rf_num_points * sizeof (float));
	
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x_q), rf_points_x_q, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y_q), rf_points_y_q, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w_q), rf_points_w_q, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g_q), rf_points_g_q, rf_num_points * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));
	//dim3 dimGrid (num_neurons);
	
	cuda_biological_gabor_right  <<< dimGrid , dimBlock >>>  (d_s_r,d_s_r_q,
				 d_image_vector, 
				 wi,hi,num_neurons, 
				 rf_num_points, 
				 global_factor, 
				 c_delta_area,teste,wo,ho,
				 min_dog,min_dog_q);


	return;
}
__global__
void cuda_translate_filter(int *image,int *image_output,int d_g_nCurrDisparity,int size,int wo)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
		
	
	//for (int i = tid; i < size; i += offset) i = 
	if( i<size )
	{	
		int xi=i%wo + d_g_nCurrDisparity;
		int yi=i/wo;

		if (xi < 0 || xi >= wo)
				image_output[i] = 0;
			else
				image_output[i] = image[(yi*wo) + xi];
	}

}
	//	for (yo = 0; yo < ho; yo++)
	//{
	//	for (xo = 0; xo < wo; xo++)
	//	{
	//		int yi = yo;
	//		int xi = xo + g_nCurrDisparity;
	//		if (xi < 0 || xi >= wo)
	//			filter_desc->output->neuron_vector[(yo*wo) + xo].output.ival = 0;
	//		else
	//			filter_desc->output->neuron_vector[(yo*wo) + xo].output = input_nl->neuron_vector[(yi*wo) + xi].output;
	//	}
	//}

extern "C" void 
cuda_biological_gabor_nls_left (float *d_s_r, float *d_s_r_q,
				 int *d_image_vector, 
				 int wi, int hi, int num_neurons, 
				 int rf_num_points, 
				 int *rf_points_x, 
				 int *rf_points_y, 
				 int *rf_points_x_q, 
				 int *rf_points_y_q,
				 int *image_vector,
				 float *rf_points_w, 
				 float *rf_points_g, 
				 float *rf_points_w_q, 
				 float *rf_points_g_q, 
				 float global_factor, 
				 float c_delta_area, float teste,int wo, int ho,
				 float min_dog,float min_dog_q, int d_g_nCurrDisparity,
				 int *I_translated_image_left)
{
	int dimension = wi*hi;

	hipMemcpy (d_image_vector, image_vector, dimension * sizeof (int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w), rf_points_w, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, rf_num_points * sizeof (float));
	
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x_q), rf_points_x_q, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y_q), rf_points_y_q, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w_q), rf_points_w_q, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g_q), rf_points_g_q, rf_num_points * sizeof (float));

	//	for (yo = 0; yo < ho; yo++)
	//{
	//	for (xo = 0; xo < wo; xo++)
	//	{
	//		int yi = yo;
	//		int xi = xo + g_nCurrDisparity;
	//		if (xi < 0 || xi >= wo)
	//			filter_desc->output->neuron_vector[(yo*wo) + xo].output.ival = 0;
	//		else
	//			filter_desc->output->neuron_vector[(yo*wo) + xo].output = input_nl->neuron_vector[(yi*wo) + xi].output;
	//	}
	//}

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));

	cuda_translate_filter<<< dimGrid, dimBlock>>> (d_image_vector,I_translated_image_left,d_g_nCurrDisparity,num_neurons,wo);
	//dim3 dimGrid (num_neurons);
	
	cuda_biological_gabor_right  <<< dimGrid , dimBlock >>>  (d_s_r,d_s_r_q,
				 I_translated_image_left, 
				 wi,hi,num_neurons, 
				 rf_num_points, 
				 global_factor, 
				 c_delta_area,teste,wo,ho,
				 min_dog,min_dog_q);

// 	cuda_biological_gabor <<< dimGrid , dimBlock >>> (d_s_m, wi, hi, num_neurons, rf_num_points, d_image_vector,
// 								global_factor, c_delta_area, teste, wo, ho,min_dog);

	return;
}

__global__ void
cuda_biological_gabor_bigfilter_nls  (float *mt, float *s_r, float *s_r_q, float *s_l, float *s_l_q,
				 int *image_vector, 
				 int wi, int hi, int size, 
				 int rf_num_points, 
				 float global_factor, 
				 float c_delta_area, float teste,int wo, int ho,
				 float min_dog,float min_dog_q,float k,int d_g_nCurrDisparity)
{
	//int offset = gridDim.x*blockDim.x;
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
		
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//for (int i = tid; i < size; i += offset) i = 
	if( i<size )
	{	
		int xi=i%wo;
		int yi=i/wo;
		
		float result,result_q;
		cuda_bidimentional_convolution_msq (rf_num_points, image_vector, xi, yi,wi, hi, global_factor, teste,&result,&result_q);
		//if(threadIdx.x==0){
		result = c_delta_area * result  / 6.0f;
		result_q = c_delta_area * result_q  / 6.0f;
			
		//s_l[i]=result = (((result > 0.0f) && (result <= min_dog)) || ((result < 0.0f) && (result >= min_dog)))? 0.0f: result;
		//s_l_q[i]=result_q = (((result_q > 0.0f) && (result_q <= min_dog_q)) || ((result_q < 0.0f) && (result_q >= min_dog_q)))? 0.0f: result_q;
		//}
		float s_lr, s_lr_q, c_l, c_r, c_lr;

		s_lr = result + s_r[i];
		s_lr_q = result_q + s_r_q[i];

		c_l = (result * result) + (result_q * result_q);
		c_r = (s_r[i] * s_r[i]) + (s_r_q[i] * s_r_q[i]);
		c_lr = (s_lr * s_lr) + (s_lr_q * s_lr_q);

		mt[i] = c_lr / (c_l + c_r + k);


	}
}


//extern "C" void 
//cuda_biological_gabor_plus_bigfilter_nls (float *mt, float *d_s_r, float *d_s_r_q, float *d_s_l, float *d_s_l_q,
//				 int *d_image_vector, 
//				 int wi, int hi, int num_neurons, 
//				 int rf_num_points, 
//				 int *rf_points_x, 
//				 int *rf_points_y, 
//				 int *rf_points_x_q, 
//				 int *rf_points_y_q,
//				 int *image_vector,
//				 float *rf_points_w, 
//				 float *rf_points_g, 
//				 float *rf_points_w_q, 
//				 float *rf_points_g_q, 
//				 float global_factor, 
//				 float c_delta_area, float teste,int wo, int ho,
//				 float min_dog,float min_dog_q,float k)
//{
//	int dimension = wi*hi;
//
//	hipMemcpy (d_image_vector, image_vector, dimension * sizeof (int), hipMemcpyHostToDevice);
//
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, rf_num_points * sizeof (int));
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, rf_num_points * sizeof (int));
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w), rf_points_w, rf_num_points * sizeof (float));
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, rf_num_points * sizeof (float));
//	
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x_q), rf_points_x_q, rf_num_points * sizeof (int));
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y_q), rf_points_y_q, rf_num_points * sizeof (int));
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w_q), rf_points_w_q, rf_num_points * sizeof (float));
//	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g_q), rf_points_g_q, rf_num_points * sizeof (float));
//
//	dim3 dimBlock (BLOCKSIZE);
//	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));
//	//dim3 dimGrid (num_neurons);
//	
//	
//	cuda_biological_gabor_bigfilter_nls  <<< dimGrid , dimBlock >>>  (mt, d_s_r, d_s_r_q, d_s_l, d_s_l_q,
//				 d_image_vector, 
//				 wi,hi,num_neurons, 
//				 rf_num_points, 
//				 global_factor, 
//				 c_delta_area,teste,wo,ho,
//				 min_dog,min_dog_q,k);
//
//// 	cuda_biological_gabor <<< dimGrid , dimBlock >>> (d_s_m, wi, hi, num_neurons, rf_num_points, d_image_vector,
//// 								global_factor, c_delta_area, teste, wo, ho,min_dog);
//
//	return;
//}



// Kernel CUDA_BIOLOGICAL_GABOR
__global__ void
cuda_biological_gabor (float *s_m,  int wi, int hi, int size, int rf_num_points, int *image_vector,
			float global_factor, float c_delta_area, float teste, int wo, int ho,float min_dog)
{
	//int offset = gridDim.x*blockDim.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	
	//for (int i = tid; i < size; i += offset) i = 
	if( i<size )
	{	
		int xi=i%wo;
		int yi=i/wo;
		float result = c_delta_area * cuda_bidimentional_convolution (rf_num_points, image_vector, xi, yi,wi, hi, global_factor, teste) / 6.0f;
									
		s_m[i] = (((result > 0.0f) && (result <= min_dog)) || ((result < 0.0f) && (result >= min_dog)))? 0.0f: result;
	}
}

// O filtro CUDA_BIOLOGICAL_GABOR_NLS eh a chamada de kernel do filtro V1_MT
extern "C" void 
cuda_biological_gabor_nls (float *d_s_m,  int *d_image_vector, int wi, int hi,
				int num_neurons, int rf_num_points, int *rf_points_x, int *rf_points_y, int *image_vector,
				float *rf_points_w, float *rf_points_g, float global_factor, float c_delta_area, float teste,int wo, int ho,float min_dog)
{
	int dimension = wi*hi;

	hipMemcpy (d_image_vector, image_vector, dimension * sizeof (int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w), rf_points_w, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, rf_num_points * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));

	cuda_biological_gabor <<< dimGrid , dimBlock >>> (d_s_m, wi, hi, num_neurons, rf_num_points, d_image_vector,
								global_factor, c_delta_area, teste, wo, ho,min_dog);

	return;
}


// O filtro CUDA_BIOLOGICAL_GABOR_NLS_OLD so pode ser chamado individualmente, na estrutura original
extern "C" void 
cuda_biological_gabor_nls_old (float *s_m, int *xi, int *yi, int wi, int hi, int num_neurons, int rf_num_points,
				int *rf_points_x, int *rf_points_y, int *image_vector, float *rf_points_w, float *rf_points_g,
				float global_factor, float c_delta_area, float teste)
{
	int  *d_xi, *d_yi, *d_image_vector;
	float *d_s_m;
	int dimension = wi*hi;

	hipMalloc ((void **) &d_image_vector, dimension * sizeof (int));
	hipMalloc ((void **) &d_xi, num_neurons * sizeof (int));
	hipMalloc ((void **) &d_yi, num_neurons * sizeof (int));

	hipMemcpy (d_image_vector, image_vector, dimension * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy (d_xi, xi, num_neurons * sizeof (int), hipMemcpyHostToDevice);
	hipMemcpy (d_yi, yi, num_neurons * sizeof (int), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, rf_num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_w), rf_points_w, rf_num_points * sizeof (float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, rf_num_points * sizeof (float));

	hipMalloc ((void **) &d_s_m, num_neurons * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid (num_neurons);

// 	cuda_biological_gabor <<< dimGrid , dimBlock >>> (d_s_m, d_xi, d_yi, wi, hi, num_neurons, rf_num_points, d_image_vector,
// 								global_factor, c_delta_area, teste);

	hipMemcpy (s_m, d_s_m, num_neurons * sizeof (float), hipMemcpyDeviceToHost);

	hipFree (d_s_m);
	hipFree (d_image_vector);
	hipFree (d_xi);
	hipFree (d_yi);

	return;
}



/*!
*********************************************************************************
* Function: cuda_add_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_ADD
__global__ void
cuda_add (float *d_C, float *d_A, float *d_B, int size)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < size )
		d_C[idx] = d_A[idx] + d_B[idx];

}

// O filtro CUDA_ADD_NLS so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_add_nls (NEURON *C, NEURON *A, NEURON *B, int size)
{

	float *h_A, *h_B, *h_C;
	float *d_A, *d_B, *d_C;

	h_A = (float *) malloc (size * sizeof (float));
	h_B = (float *) malloc (size * sizeof (float));
	h_C = (float *) malloc (size * sizeof (float));

	hipMalloc ((void **) &d_A, size * sizeof (float));
	hipMalloc ((void **) &d_B, size * sizeof (float));
	hipMalloc ((void **) &d_C, size * sizeof (float));

	copyneuron2floatvec (h_A, A, size);
	copyneuron2floatvec (h_B, B, size);

	hipMemcpy (d_A, h_A, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_B, h_B, size * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((size/dimBlock.x) + (!(size%dimBlock.x)?0:1));

	cuda_add <<< dimGrid , dimBlock >>> (d_C, d_A, d_B, size);

	hipMemcpy(h_C, d_C, size * sizeof (float), hipMemcpyDeviceToHost);

	copyfloatvec2neuron (C, h_C, size);

	hipFree (d_A);
	hipFree (d_B);
	hipFree (d_C);

	return;
}



/*!
*********************************************************************************
* Function: cuda_mult_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_MULT_ADD
__global__ void
cuda_mult_add (float *d_C, float *d_A, float *d_B, int size)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < size )
		d_C[idx] = (d_A[idx] * d_A[idx]) + (d_B[idx] * d_B[idx]);

}

// O filtro CUDA_MULT_NLS so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_mult_nls (NEURON *C, NEURON *A, NEURON *B, int size)
{
	float *h_A, *h_B, *h_C;
	float *d_A, *d_B, *d_C;

	h_A = (float *) malloc (size * sizeof (float));
	h_B = (float *) malloc (size * sizeof (float));
	h_C = (float *) malloc (size * sizeof (float));

	hipMalloc ((void **) &d_A, size * sizeof (float));
	hipMalloc ((void **) &d_B, size * sizeof (float));
	hipMalloc ((void **) &d_C, size * sizeof (float));

	copyneuron2floatvec (h_A, A, size);
	copyneuron2floatvec (h_B, B, size);

	hipMemcpy (d_A, h_A, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_B, h_B, size * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((size/dimBlock.x) + (!(size%dimBlock.x)?0:1));

	cuda_mult_add <<< dimGrid , dimBlock >>> (d_C, d_A, d_B, size);

	hipMemcpy(h_C, d_C, size * sizeof (float), hipMemcpyDeviceToHost);

	copyfloatvec2neuron (C, h_C, size);

	hipFree (d_A);
	hipFree (d_B);
	hipFree (d_C);

	return;
}



/*!
*********************************************************************************
* Function: cuda_div_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_DIV
__global__ void
cuda_div (float *d_D, float *d_A, float *d_B, float *d_C, int size, float k_param)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < size )
		d_D[idx] = d_C[idx] / (d_A[idx] + d_B[idx] + k_param);

}

// O filtro CUDA_DIV_NLS so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_div_nls (NEURON *D, NEURON *A, NEURON *B, NEURON *C, int size, float k_param)
{
	float *h_A, *h_B, *h_C, *h_D;
	float *d_A, *d_B, *d_C, *d_D;

	h_A = (float *) malloc (size * sizeof (float));
	h_B = (float *) malloc (size * sizeof (float));
	h_C = (float *) malloc (size * sizeof (float));
	h_D = (float *) malloc (size * sizeof (float));

	hipMalloc ((void **) &d_A, size * sizeof (float));
	hipMalloc ((void **) &d_B, size * sizeof (float));
	hipMalloc ((void **) &d_C, size * sizeof (float));
	hipMalloc ((void **) &d_D, size * sizeof (float));

	copyneuron2floatvec (h_A, A, size);
	copyneuron2floatvec (h_B, B, size);
	copyneuron2floatvec (h_C, C, size);

	hipMemcpy (d_A, h_A, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_B, h_B, size * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_C, h_C, size * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((size/dimBlock.x) + (!(size%dimBlock.x)?0:1));

	cuda_div <<< dimGrid , dimBlock >>> (d_D, d_A, d_B, d_C, size, k_param);

	hipMemcpy (h_D, d_D, size * sizeof (float), hipMemcpyDeviceToHost);

	copyfloatvec2neuron (D, h_D, size);

	hipFree (d_A);
	hipFree (d_B);
	hipFree (d_C);
	hipFree (d_D);

	return;
}



/*!
*********************************************************************************
* Function: cuda_bigfilter_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Kernel CUDA_BIGFILTER
__global__ void
cuda_bigfilter (float *mt, float *s_r, float *s_r_q, float *s_l, float *s_l_q, int num_neurons, float k)
{
	float s_lr, s_lr_q, c_l, c_r, c_lr;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < num_neurons )
	{

		s_lr = s_l[idx] + s_r[idx];
		s_lr_q = s_l_q[idx] + s_r_q[idx];

		c_l = (s_l[idx] * s_l[idx]) + (s_l_q[idx] * s_l_q[idx]);
		c_r = (s_r[idx] * s_r[idx]) + (s_r_q[idx] * s_r_q[idx]);
		c_lr = (s_lr * s_lr) + (s_lr_q * s_lr_q);

		mt[idx] = c_lr / (c_l + c_r + k);
	}
}

// O filtro CUDA_BIGFILTER_NLS eh a chamada de kernel do filtro V1_MT
extern "C" void
cuda_bigfilter_nls (float *d_mt, float *d_s_r, float *d_s_r_q, float *d_s_l, float *d_s_l_q, int num_neurons, float k)
{
	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));

	cuda_bigfilter <<< dimGrid , dimBlock >>> (d_mt, d_s_r, d_s_r_q, d_s_l, d_s_l_q, num_neurons, k);

	return;
}

// O filtro CUDA_BIGFILTER_NLS_OLD so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_bigfilter_nls_old (float *mt, float *s_r, float *s_r_q, float *s_l, float *s_l_q, int num_neurons, float k)
{
	float *d_mt, *d_s_r, *d_s_r_q, *d_s_l, *d_s_l_q;

	hipMalloc ((void **) &d_mt, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_r, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_r_q, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_l, num_neurons * sizeof (float));
	hipMalloc ((void **) &d_s_l_q, num_neurons * sizeof (float));

	hipMemcpy (d_s_r, s_r, num_neurons * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_s_r_q, s_r_q, num_neurons * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_s_l, s_l, num_neurons * sizeof (float), hipMemcpyHostToDevice);
	hipMemcpy (d_s_l_q, s_l_q, num_neurons * sizeof (float), hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid ((num_neurons/dimBlock.x) + (!(num_neurons%dimBlock.x)?0:1));

	cuda_bigfilter <<< dimGrid , dimBlock >>> (d_mt, d_s_r, d_s_r_q, d_s_l, d_s_l_q, num_neurons, k);

	hipMemcpy (mt, d_mt, num_neurons * sizeof (float), hipMemcpyDeviceToHost);

	hipFree (d_mt);
	hipFree (d_s_r);
	hipFree (d_s_r_q);
	hipFree (d_s_l);
	hipFree (d_s_l_q);

	return;
}



/*!
*********************************************************************************
* Function: cuda_gaussian_nls
* Description:
* Inputs:
* Output:
*********************************************************************************
*/

// Device CUDA_GAUSSIAN
// __device__ float
// cuda_apply_gaussian_kernel (float *mt, int num_points, int x0, int y0, int wi, int hi)
// {
// 	int i, j, k;
// 	int xr, yr;
// 	float fltWeight;
// 
// 	__shared__ float fltWeightSum[BLOCKSIZE];
// 	__shared__ float fltResult[BLOCKSIZE];
// 
// 	fltWeightSum[threadIdx.x] = 0.0f;
// 	fltResult[threadIdx.x] = 0.0f;
// 
// 	for (i = threadIdx.x; i < num_points; i += blockDim.x)
// 	{
// 		xr = x0 + d_rf_points_x[i];
// 		yr = y0 + d_rf_points_y[i];
// 
// 		if ((xr < 0) || (xr >= wi) || (yr < 0) || (yr >= hi))
// 			continue;
// 
// 		fltWeightSum[threadIdx.x] += fltWeight = d_rf_points_g[i];
// 		fltResult[threadIdx.x] += fltWeight * mt[yr * wi + xr];
// 	}
// 
// 	for (k = BLOCKSIZE / 2; k > 0; k >>=1)
// 	{
// 		__syncthreads();
// 
// 		for (j = threadIdx.x; j < k; j += blockDim.x)
// 		{
// 			fltWeightSum[j] += fltWeightSum[k + j];
// 			fltResult[j] += fltResult[k + j];
// 		}
// 	}
// 
// 	__syncthreads();
// 
// 	return ((fltWeightSum[0] != .0f) ? fltResult[0] / fltWeightSum[0] : .0f);
// }

__device__ float
cuda_apply_gaussian_kernel (float *mt, int num_points, int x0, int y0, int wi, int hi)
{
	int i, j, k;
	int xr, yr;
	float fltWeight;

	__shared__ float fltWeightSum[BLOCKSIZE];
	__shared__ float fltResult[BLOCKSIZE];

	fltWeightSum[threadIdx.x] = 0.0f;
	fltResult[threadIdx.x] = 0.0f;

	for (i = threadIdx.x; i < num_points; i += blockDim.x)
	{
		xr = x0 + d_rf_points_x[i];
		yr = y0 + d_rf_points_y[i];

		if ((xr < 0) || (xr >= wi) || (yr < 0) || (yr >= hi))
			continue;

		fltWeightSum[threadIdx.x] += fltWeight = d_rf_points_g[i];
		fltResult[threadIdx.x] += fltWeight * mt[yr * wi + xr];
	}

	j=threadIdx.x;
	for (k = BLOCKSIZE / 2; k > 0; k >>=1)
	{
		__syncthreads();

		if( j < k )
		{
			fltWeightSum[j] += fltWeightSum[k + j];
			fltResult[j] += fltResult[k + j];
		}
	}

	__syncthreads();

	return ((fltWeightSum[0] != 0.0f) ? fltResult[0] / fltWeightSum[0] : 0.0f);
}

// Kernel CUDA_GAUSSIAN
__global__ void
cuda_gaussian (float *mt_gaussian, float *mt, int num_points, int wo, int ho, int wi, int hi)
{
	for (int i = blockIdx.x; i < wo; i += gridDim.x)
	{
		for (int j = blockIdx.y; j < ho; j += gridDim.y)
		{
			mt_gaussian[j * wo + i] = cuda_apply_gaussian_kernel (mt, num_points, i, j, wo, ho);
		}
	}
}

// O filtro CUDA_GAUSSIAN_NLS eh a chamada de kernel do filtro V1_MT
extern "C" void
cuda_gaussian_nls (float *mt_gaussian, float *d_mt_gaussian, float *d_mt, int *rf_points_x, int *rf_points_y, 
		  float *rf_points_g, int num_points, int wo, int ho, int wi, int hi)
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, num_points * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid (wo,ho);

	cuda_gaussian <<< dimGrid , dimBlock >>> (d_mt_gaussian, d_mt, num_points, wo, ho, wi, hi);

	hipMemcpy (mt_gaussian, d_mt_gaussian, wo*ho * sizeof (float), hipMemcpyDeviceToHost);
}

// O filtro CUDA_GAUSSIAN_NLS_OLD so pode ser chamado individualmente, na estrutura original
extern "C" void
cuda_gaussian_nls_old (float *mt_gaussian, float *mt, int *rf_points_x, int *rf_points_y, 
		  float *rf_points_g, int num_points, int wo, int ho, int wi, int hi)
{
	float *d_mt, *d_mt_gaussian;

	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_x), rf_points_x, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_y), rf_points_y, num_points * sizeof (int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_rf_points_g), rf_points_g, num_points * sizeof (float));

	hipMalloc ((void **) &d_mt, wi*hi * sizeof (float));
	hipMemcpy (d_mt, mt, wi*hi * sizeof (float), hipMemcpyHostToDevice);

	hipMalloc ((void **) &d_mt_gaussian, wo*ho * sizeof (float));

	dim3 dimBlock (BLOCKSIZE);
	dim3 dimGrid (wo,ho);

	cuda_gaussian <<< dimGrid , dimBlock >>> (d_mt_gaussian, d_mt, num_points, wo, ho, wi, hi);

	hipMemcpy (mt_gaussian, d_mt_gaussian, wo*ho * sizeof (float), hipMemcpyDeviceToHost);

	hipFree (d_mt_gaussian);
	hipFree (d_mt);
}
